#include "hip/hip_runtime.h"
#include "cuda_tensor_ops.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA_ERROR(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", \
                __FILE__, __LINE__, err, hipGetErrorString(err), #call); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

template <int BLOCK_SIZE>
__global__ void gemm_kernel(const float* A, const float* B, float* C,
                            int M, int N, int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Coordinate in C matrix
    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.0f;

    for (int i = 0; i < K; i += BLOCK_SIZE) {
        // Load tile from A and B into shared memory
        if (row < M && (i + tx) < K)
            As[ty][tx] = A[row * K + i + tx];
        else
            As[ty][tx] = 0.0f;

        if ((i + ty) < K && col < N)
            Bs[ty][tx] = B[(i + ty) * N + col];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < BLOCK_SIZE; ++k)
            sum += As[ty][k] * Bs[k][tx];

        __syncthreads();
    }

    // Write result to C
    if (row < M && col < N)
        C[row * N + col] = sum;
}

template <int BLOCK_SIZE, int RANK>
__global__ void gemm_lora_kernel(const float* A, const float* B, float* C,
                                 int M, int N, int K,
                                 const float* lora_A, const float* lora_B,
                                 float lora_alpha) {
    // Main GEMM computation same as above...

    // Add LoRA contribution
    if (lora_A && lora_B) {
        float lora_sum = 0.0f;
        for (int r = 0; r < RANK; ++r) {
            lora_sum += lora_A[row * RANK + r] * lora_B[r * N + col];
        }
        sum += lora_alpha * lora_sum;
    }

    // Write result...
}

void cuda_gemm(const float* A, const float* B, float* C,
               int M, int N, int K,
               const float* lora_A,
               const float* lora_B,
               int lora_rank,
               float lora_alpha,
               hipStream_t stream) {
    const int BLOCK_SIZE = 16;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
              (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    if (lora_A && lora_B && lora_rank > 0) {
        switch (lora_rank) {
            case 8:  gemm_lora_kernel<BLOCK_SIZE, 8><<<grid, block, 0, stream>>>(...); break;
            case 16: gemm_lora_kernel<BLOCK_SIZE,16><<<grid, block, 0, stream>>>(...); break;
            default: throw std::invalid_argument("Unsupported LoRA rank");
        }
    } else {
        gemm_kernel<BLOCK_SIZE><<<grid, block, 0, stream>>>(A, B, C, M, N, K);
    }
    CHECK_CUDA_ERROR(hipGetLastError());
}
